
#include <hip/hip_runtime.h>
__host__ __device__
int get_raw(int idx, int K_k) {
	return idx / K_k;
}

__host__ __device__
int get_col(int idx, int K_k) {
	return idx % K_k;
}

__host__ __device__
void get_mul(double* res, double* a, double* b, int idx, int K_m, int K_n, int K_k ) {
	int m = get_raw(idx, K_k);
	int k = get_col(idx, K_k);

	for (int n = 0; n < K_n; n++)
		res[idx] = a[m * K_n + n] * b[n * K_k + k];
}

__global__
void mmul(double* a, double* b, double* res, int K_m, int K_n, int K_k) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < K_m * K_k)
		get_mul(res, a, b, id, K_m, K_n, K_k);

}
